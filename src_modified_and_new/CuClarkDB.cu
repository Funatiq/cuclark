#include "hip/hip_runtime.h"
/*
   CuCLARK, CLARK for CUDA-enabled GPUs.
   Copyright 2016, Robin Kobus <rkobus@students.uni-mainz.de>
   
   based on CLARK version 1.1.3, CLAssifier based on Reduced K-mers.
   Copyright 2013-2016, Rachid Ounit <rouni001@cs.ucr.edu>


   This program is free software: you can redistribute it and/or modify
   it under the terms of the GNU General Public License as published by
   the Free Software Foundation, either version 3 of the License, or
   (at your option) any later version.

   This program is distributed in the hope that it will be useful,
   but WITHOUT ANY WARRANTY; without even the implied warranty of
   MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
   GNU General Public License for more details.

   You should have received a copy of the GNU General Public License
   along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */

/*
 * @author: Robin Kobus, masters student at Institute of Computer Science, JGU Mainz
 * @project: CuCLARK, Metagenomic Classification with CUDA-enabled GPUs
 * 
 * New file:
 * New database structure and GPU handling for database and queries.
 * Includes kernels to query batches of reads, merge results and find best results.
 */
 
#include "CuClarkDB.cuh"

#include <iostream>
#include <fstream>
#include <iterator>
//~ #include <numeric>	// partial_sum
#include <cstring>	// memcpy

// for debugging prints
//~ #include <inttypes.h>	// PRIu64 print makro
//~ #include <bitset>		// print kmer container

#define CUERR {														\
	hipError_t err;												\
	if ((err = hipGetLastError()) != hipSuccess)					\
	{																\
		std::cerr << "CUERR '" << hipGetErrorString(err) << "' in "\
				  << __FILE__ << ", line " << __LINE__ << "\n";		\
		exit(1);													\
	}																\
}

#define CUMEMERR {													\
	if (hipGetLastError()== hipErrorOutOfMemory)				\
	{																\
		std::cerr << "ERROR: Out of GPU memory.\n"					\
				  << "Please increase the number of batches "		\
				  << "(-b <numberofbatches>).\n";					\
		exit(1);													\
	}																\
}

// forward declaration
template <typename HKMERr>
__device__ bool queryElement (const uint8_t& k, const uint64_t& _ikmer,
			uint32_t* d_bucketPointers, HKMERr* d_keys, ILBL* d_labels,
			uint32_t dbPartStart, uint32_t dbPartEnd,
			//~ uint8_t dbParts, uint8_t dbPart,
			ILBL& _returnLabel);
template <typename HKMERr>
__global__ void queryKernel (uint8_t k,
			uint32_t* readsPointer, CONTAINER* readsInContainers,
			uint32_t* bucketPointers, HKMERr* keys, ILBL* labels,
			uint32_t dbPartStart, uint32_t dbPartEnd,
			//~ uint8_t dbParts, uint8_t dbPart,
			RESULTS* results, size_t pitch, size_t numTargets);
__global__ void mergeKernel (RESULTS* resultA, RESULTS* resultB, size_t pitch, size_t numReads, RESULTS* results);
__global__ void resultKernel (RESULTS* scores, size_t spitch, size_t numReads, RESULTS* results, size_t rpitch);

/**
 * Constructor:
 * Initialize variables, find CUDA devices
 */	
template <typename HKMERr>
CuClarkDB<HKMERr>::CuClarkDB(const size_t _numDevices, const uint8_t _k, const size_t _numBatches, const size_t _numTargets)
							: m_k(_k),m_numTargets(_numTargets),m_numBatches(_numBatches)
{	
	m_numReads.resize(m_numBatches);
	m_sizeReadsPointer.resize(m_numBatches);
	m_sizeReadsInContainers.resize(m_numBatches);
	
	h_readsPointer.resize(m_numBatches);
	h_readsInContainers.resize(m_numBatches);
	
	h_results.resize(m_numBatches);
	h_resultsFinal.resize(m_numBatches);
	
	m_batchFinishedEvents.resize(m_numBatches);
	for(int i=0; i <m_numBatches; i++)
	{
		hipEventCreateWithFlags(&m_batchFinishedEvents[i],hipEventDisableTiming);
	}
	
	m_numDevices = 0;
	m_dbPartsPerDevice = 1;
	
	// cf. CUDA samples/0_Simple/simpleP2P
	std::cerr << "Checking for CUDA devices: ";
	hipGetDeviceCount(&m_numDevices);
	CUERR
	if (m_numDevices > 0)
		std::cerr << m_numDevices << " device(s) found.\n";
	else
	{
		std::cerr << "No CUDA devices found. Abort.\n";
		exit(1);
	}
	
	std::vector<hipDeviceProp_t> prop(m_numDevices);
	for(int i=0; i<m_numDevices; i++)
	{
		hipGetDeviceProperties(&prop[i], i);
		CUERR
		std::cerr << "Device " << i << " = " << prop[i].name << "\n";
	}
	
	if (m_numDevices < _numDevices)
	{
		std::cerr << _numDevices << " CUDA devices requested. Insufficient devices found. Abort.\n";
		exit(1);
	}
	
	if (_numDevices > 0)
	{
		std::cerr << "Using " << _numDevices << " CUDA devices as requested.\n";
		m_numDevices = _numDevices;
	}
	
	for(int i=0; i<m_numDevices; i++)
	{
		if(strcmp(prop[i].name,"GeForce GTX TITAN X") == 0)
			m_dbPartsPerDevice = DBPARTSPERDEVICE;
	}
	
	std::vector<int>	gpuid(m_numDevices);
	int gpu_count = 0;
	m_memSizes.resize(m_numDevices*m_dbPartsPerDevice);
	
	for(int i=0; i<m_numDevices; i++)
	{
		// locate devices capable of Peer-to-Peer
		if(prop[i].major >= 2)
		{
			gpuid[gpu_count++] = i;
			//~ std::cerr << "Device " << i << " = " << prop[i].name << " is capable of P2P.\n";
			//~ printf("> GPU%d = \"%15s\" is capable of Peer-to-Peer (P2P)\n", i, prop[i].name);
		}
		

		size_t freeMem, totalMem;
		hipSetDevice(i);
		hipMemGetInfo(&freeMem, &totalMem);
		CUERR
		//~ std::cerr << "Device " << i << " free: " << freeMem/1000000
									//~ << " total: " << totalMem/1000000
									//~ << " global: " << prop[i].totalGlobalMem/1000000 
									//~ << std::endl;
		if (freeMem < 1000000000)
		{
			std::cerr << "Device " << i << " has less than 1GB of free memory. Abort.\n";
			exit(1);
		}
		
		freeMem -= RESERVED;
		for (int j=0; j<m_dbPartsPerDevice; ++j)
		{
			m_memSizes[m_dbPartsPerDevice*i+j] = freeMem/m_dbPartsPerDevice + (i+j>0 ? m_memSizes[m_dbPartsPerDevice*i+j-1] : 0);		
		}
	}
	
	int can_access_peer;
	// check all the combinations of supported P2P GPUs
    for (int i = 0; i < gpu_count; i++)
    {
        for (int j = i+1; j < gpu_count; j++)
        {
            if (gpuid[i] == gpuid[j])
            {
                continue;
            }
            hipDeviceCanAccessPeer(&can_access_peer, gpuid[i], gpuid[j]);
            CUERR
            
            if(can_access_peer)
            {
				std::cerr << "Enabling peer access between devices " << gpuid[i] << " and " << gpuid[j] << "\n";
				hipSetDevice(gpuid[i]);
				hipDeviceEnablePeerAccess(gpuid[j], 0);
				CUERR
				hipSetDevice(gpuid[j]);
				hipDeviceEnablePeerAccess(gpuid[i], 0);
				CUERR
			}
        }
    }
    
    // pointers for each device
    d_bucketPointers.resize(m_numDevices*m_dbPartsPerDevice);
	d_keys.resize(m_numDevices*m_dbPartsPerDevice);
	d_labels.resize(m_numDevices*m_dbPartsPerDevice);
	
	d_readsPointer.resize(m_numDevices);
	d_readsInContainers.resize(m_numDevices);
	
	d_results.resize(m_numDevices);

	for (int i=0; i<m_numDevices; i++)
	{
		if (m_dbPartsPerDevice > 1)
		{// results for each part, +1 for merging
			d_results[i].resize(m_dbPartsPerDevice+1);
		}
		else
		{// 3 results on every other device for merging
			if (i%2 == 0)
				d_results[i].resize(3);
			else
				d_results[i].resize(1);
		}		
	}
	
	d_resultsFinal.resize(m_numDevices);
}

/**
 * Destructor:
 * Free memory allocations
 */
template <typename HKMERr>
CuClarkDB<HKMERr>::~CuClarkDB() 
{
	for (int i=0; i<m_dbParts; i++)
	{		
		hipHostFree(h_bucketPointers[i]);
		hipHostFree(h_keys[i]);
		hipHostFree(h_labels[i]);
	}

	for(int i=0; i<m_numDevices; i++)
	{
		hipSetDevice(i);
		
		for(int j=0; j<m_dbPartsPerDevice; ++j)
		{
			int index = m_dbPartsPerDevice*i+j;
			hipFree(d_bucketPointers[index]);
			hipFree(d_keys[index]);
			hipFree(d_labels[index]);
			//~ CUERR
		}
	}
	
	for(int i=0; i<m_batchFinishedEvents.size(); i++)
		hipEventDestroy(m_batchFinishedEvents[i]);
	//~ CUERR
	
	for(int i=0; i<m_numDevices; i++)
	{
		hipSetDevice(i);
		CUERR
		hipDeviceReset();
	}
}

/**
 * Reset memory allocations before new input file.
 */
template <typename HKMERr>
void CuClarkDB<HKMERr>::free()
{
	//~ hipSetDevice(0);
	for (int i=0; i<m_numBatches; i++)
	{
		hipHostFree(h_readsPointer[i]);
		hipHostFree(h_readsInContainers[i]);
		CUERR
	}
	
	hipHostFree(h_results[0]);
	hipHostFree(h_resultsFinal[0]);
	CUERR	
	
	for(int i=0; i<m_numDevices; i++)
	{
		hipSetDevice(i);
		hipDeviceSynchronize();
		CUERR
		
		hipFree(d_readsPointer[i]);
		hipFree(d_readsInContainers[i]);
		CUERR
		
		for (int j=0; j<d_results[i].size(); j++)
			hipFree(d_results[i][j]);
		CUERR
		
		hipFree(d_resultsFinal[i]);
		CUERR
	}

	//~ hipSetDevice(0);
	//~ if(d_resultsFinal)
	//~ {
		//~ hipFree(d_resultsFinal);
	//~ }
	//~ CUERR
}

/**
 * Allocate pinned host memory and device memory for batch data and results.
 */
template <typename HKMERr>
size_t CuClarkDB<HKMERr>::malloc(size_t _numReads,
						size_t _maxReads, size_t _maxReadsInContainers,
						std::vector<ITYPE>& _indexBatches,
						RESULTS* &_fullResults,	size_t _resultRowSize,
						RESULTS* &_finalResults, size_t _finalResultsRowSize,
						bool _isExtended,
						std::vector<uint32_t*>& _readsPointer,
						std::vector<CONTAINER*>& _readsInCon)
{
	m_sizeResultRow			= _resultRowSize*sizeof(RESULTS);
	m_sizeResultFinalRow	= _finalResultsRowSize*sizeof(RESULTS);
	
	// set parameters for query kernel
	size_t numWarps = 2;
	size_t warpSize = 32;
	m_threadsPerBlock_queryKernel = warpSize*numWarps;
	//~ size_t numTargets = (((m_numTargets-1) / threadsPerBlock) +1) *threadsPerBlock;	
	
	// hit counters in shared
	m_sharedSize_queryKernel = ((m_numTargets-1)/2+1)*2*sizeof(uint16_t);
	// containers in shared
	size_t sharedMemPerWarp = ((m_k+warpSize-2)/(sizeof(CONTAINER)*4)+1)*sizeof(CONTAINER);
	// result buffer in shared
	if (m_sizeResultRow > sharedMemPerWarp) sharedMemPerWarp = m_sizeResultRow;
	m_sharedSize_queryKernel += sharedMemPerWarp*numWarps;

	//~ std::cerr << "Required Shared Memory per block: \t" << m_sharedSize_queryKernel/1000.0 << " KB\n";
	
	size_t total = 0;
	
	size_t maxReadsPointer = _maxReads+1;
	size_t sizeReadsPointer			= maxReadsPointer*sizeof(uint32_t),
		   sizeReadsInContainers	= _maxReadsInContainers*sizeof(CONTAINER);
		   //~ sizeResultRow 			= _resultRowSize*sizeof(RESULTS);
	
	for (int i=0; i<m_numBatches; i++)
	{
		hipHostMalloc(&h_readsPointer[i], sizeReadsPointer);
		hipHostMalloc(&h_readsInContainers[i], sizeReadsInContainers);
		CUERR
	}
	_readsPointer = h_readsPointer;
	_readsInCon = h_readsInContainers;
	
	for (int i=0; i<m_numDevices; i++)
	{
		hipSetDevice(i);
		
		// allocate space for reads on each device
		hipMalloc(&d_readsPointer[i], sizeReadsPointer);
		CUMEMERR
		hipMalloc(&d_readsInContainers[i], sizeReadsInContainers);
		CUMEMERR	

		// allocate space for each partitial result & merging
		for (int j=0; j<d_results[i].size(); j++)
			hipMalloc(&d_results[i][j], m_sizeResultRow*_maxReads);
		//~ std::cerr << i << " allocated\n";
		CUMEMERR
	}
	
	total += sizeReadsPointer;
	total += sizeReadsInContainers;
	total += m_sizeResultRow*_maxReads*d_results[0].size();
	
	// allocate space to store full results on host
	if (m_dbParts > 1 || _isExtended)
	{
		//~ std::cerr << "extra allocation\n";
		hipHostMalloc(&_fullResults, m_sizeResultRow*_numReads);
		CUERR
		//~ std::cerr << "Full result size:\t" << m_sizeResultRow*_numReads/1000/1000.0 << " MB\n";
	}
	
	// allocate space for final result & async copy to host
	if (_finalResultsRowSize > 0)
	{
		//~ hipSetDevice(0);
		hipHostMalloc(&_finalResults, m_sizeResultFinalRow*_numReads);
		CUERR
		
		for (int i=0; i<m_numDevices; i++)
		{
			hipSetDevice(i);
			
			hipMalloc	  (&d_resultsFinal[i],  m_sizeResultFinalRow*_maxReads);
			CUMEMERR
		}
		
		total += m_sizeResultFinalRow*_maxReads;
	}
	//~ std::cerr << "Final result size:\t" << m_sizeResultFinalRow*_numReads/1000/1000.0 << " MB\n";
	
	for (int i=0; i<m_numBatches; i++)
	{
		h_results[i] = _fullResults+_resultRowSize*_indexBatches[i];	
		h_resultsFinal[i] = _finalResults+_finalResultsRowSize*_indexBatches[i];
	}
	
	return total;
}

/** 
 * Synchronize all CUDA devices and check for errors.
 */
template <typename HKMERr>
bool CuClarkDB<HKMERr>::sync()
{
	for (int i=0; i<m_numDevices; i++)
	{
		hipSetDevice(i);
		hipDeviceSynchronize();
		CUERR
	}
	return true;
}

/** 
 * Wait for GPUs to finish a batch.
 */
template <typename HKMERr>
bool CuClarkDB<HKMERr>::waitForBatch(size_t batchId)
{
	hipEventSynchronize(m_batchFinishedEvents[batchId]);
	return hipGetLastError() == hipSuccess;
}

/** 
 * Check if GPUs have already finished a batch.
 */
template <typename HKMERr>
bool CuClarkDB<HKMERr>::checkBatch(size_t batchId)
{
	hipEventQuery(m_batchFinishedEvents[batchId]);
	return hipGetLastError() == hipSuccess;
}

/** 
 * Read database from files to pinned host memory,
 * divide into parts according to CUDA device memory.
 * cf. hashTable_hh read
 */
template <typename HKMERr>
bool CuClarkDB<HKMERr>::read (const char * _filename, size_t& _fileSize, size_t& _dbParts, const ITYPE& _modCollision, const bool& _isfastLoadingRequested)
{

	char * file_sze = (char*) calloc(strlen(_filename)+4,sizeof(char));
	char * file_key = (char*) calloc(strlen(_filename)+4,sizeof(char));
	char * file_lbl = (char*) calloc(strlen(_filename)+4,sizeof(char));

	sprintf(file_sze, "%s.sz", _filename);
	sprintf(file_key, "%s.ky", _filename);
	sprintf(file_lbl, "%s.lb", _filename);

	std::ifstream ifs_sze;
	std::ifstream ifs_key;
	std::ifstream ifs_lbl;

//~ #define BUFFERSIZE 64001

	//~ char buffer_sze[BUFFERSIZE];
	//~ ifs_sze.rdbuf()->pubsetbuf(buffer_sze,BUFFERSIZE);
	//~ char buffer_key[BUFFERSIZE];
	//~ ifs_key.rdbuf()->pubsetbuf(buffer_key,BUFFERSIZE);
	//~ char buffer_lbl[BUFFERSIZE];	
	//~ ifs_lbl.rdbuf()->pubsetbuf(buffer_lbl,BUFFERSIZE);
	
	ifs_sze.open(file_sze, std::ios::binary);
	ifs_key.open(file_key, std::ios::binary);
	ifs_lbl.open(file_lbl, std::ios::binary);
	
	if (!ifs_sze.is_open())
	{	std::cerr << "Failed to open " << file_sze << std::endl; return false;	}
	if (!ifs_key.is_open())
	{	std::cerr << "Failed to open " << file_key << std::endl; return false;   }
	if (!ifs_lbl.is_open())
	{	std::cerr << "Failed to open " << file_lbl << std::endl; return false;   }
	
	bool allCollision = _modCollision <= 1;
	
	/// read bucket sizes
	std::vector<uint8_t>	bucketSizes;
	bucketSizes.resize(HTSIZE);
	
	ifs_sze.read((char*) &bucketSizes[0], HTSIZE);

	uint64_t nbElements = 0;
	uint32_t nbNonZeroBuckets = 0;
	//~ size_t bucketSizeMax = 0;
	std::vector<uint8_t>	choice(HTSIZE,0);
	
	// choose buckets and count chosen elements
	for (uint32_t i = 0; i < HTSIZE; i++)
	{
		if (bucketSizes[i] > 0)
		{
			nbNonZeroBuckets++;
			// 2 = keep, 1 = skip, 0 = empty
			choice[i] = (allCollision || (nbNonZeroBuckets % _modCollision)== 0) ? 2: 1;
			if (choice[i] == 2)
				nbElements += bucketSizes[i];
				
			//~ if (bucketSizeMax < bucketSizes[i])
				//~ bucketSizeMax = bucketSizes[i];
		}
	}
	
	//~ std::cerr << "AVG bucket size: " << (float)nbElements/HTSIZE
			  //~ << ", MAX bucket size: " << (int)bucketSizeMax;
	
	// calculate total size
	_fileSize = HTSIZE;	
	// bucket pointers: 8bit -> 32bit per element
	_fileSize *= sizeof(uint32_t);
	// size of keys and labels
	size_t _fileSizeKeys   = nbElements * sizeof(HKMERr);
	size_t _fileSizeLabels = nbElements * sizeof(ILBL);
	// total database size
	_fileSize = _fileSize + _fileSizeKeys + _fileSizeLabels;
	std::cerr << "Total DB size in RAM:\t" << _fileSize/1000000/1000.0 << " GB\n";
	
	/// divide into parts
	std::cerr << "Total device memory:\t" << m_memSizes.back()/1000000/1000.0 << " GB (" << m_numDevices*RESERVED/1000000 << " MB reserved)\n";
	
	size_t minParts = (nbElements/(uint32_t)-1)+1;
	//~ std::cerr << ", Need at least " << minParts << " part(s).\n";

	m_cyclesPerDevice = _fileSize / m_memSizes.back() + 1;
	m_dbParts = m_cyclesPerDevice*m_numDevices*m_dbPartsPerDevice;
	
	// adjust number of parts to prevent overflow
	if (m_dbParts < minParts)
	{
		std::cerr << "Overflow prevented.\n";
		m_cyclesPerDevice = (minParts-1)/(m_numDevices*m_dbPartsPerDevice)+1;
		m_dbParts = m_cyclesPerDevice*m_numDevices*m_dbPartsPerDevice;
	}
	
	m_cyclesToDo = m_cyclesPerDevice;
	_dbParts = m_cyclesPerDevice;
	
	// no space for merging needed if db fits on one device
	if (m_dbParts == 1)
		d_results[0].resize(1);
	
	std::cerr << "Requiring " << m_cyclesPerDevice << " loop(s).\n";

	// point to db parts in bucket list
	//~ std::vector<uint32_t> m_partPointer(m_dbParts+1, 0);		// named Pos in hashTable_hh
	m_partPointer.resize(m_dbParts+1);
	m_partPointer[0] = 0;
	for(int i = 0; i < m_dbParts ; i++)
	{
		//~ m_partPointer[i] = ((HTSIZE-1)/m_dbParts +1) * i;		
		//~ m_partPointer[i] = HTSIZE/m_dbParts * i;
		m_partPointer[1+i] = HTSIZE/m_cyclesPerDevice * (i/(m_numDevices*m_dbPartsPerDevice) + (float)m_memSizes[i%(m_numDevices*m_dbPartsPerDevice)] / m_memSizes[m_numDevices*m_dbPartsPerDevice-1]);
	}	
	m_partPointer[m_dbParts] = HTSIZE;
	
	//~ for(int i = 0; i <= m_dbParts ; i++)
	//~ {
		//~ std::cerr << "m_partPointer " << m_partPointer[i] << " \n";
	//~ }
	
	/// calculate pointers to buckets
	
	// pointers for each part
	h_bucketPointers.resize(m_dbParts);
	h_keys.resize(m_dbParts);
	h_labels.resize(m_dbParts);
	
	// sizes for each part
	m_partSize.resize(m_dbParts);
	m_partSizeKeys.resize(m_dbParts);
	m_partSizeLabels.resize(m_dbParts);
	
	/*	single part version
	std::vector<uint32_t>	h_bucketPointers;
	h_bucketPointers.resize(HTSIZE+1);
	h_bucketPointers[0] = 0;
	
	// partial_sum
	std::vector<uint8_t>::const_iterator first = bucketSizes.begin();
	std::vector<uint8_t>::const_iterator last  = bucketSizes.end();
	std::vector<uint32_t>::iterator result = h_bucketPointers.begin()+1;

	if (first!=last) {
	  uint32_t val = *first;
	  *result = val;
	  while (++first!=last) {
		val = val + *first;
		*++result = val;
	  }
	  ++result;
	}
	
	//~ std::copy(h_bucketPointers.begin(), h_bucketPointers.begin()+5,
				//~ std::ostream_iterator<unsigned>(std::cerr, " "));	          		
	//~ std::cerr << '\n';
	*/
		
	#ifdef _OPENMP
	#pragma omp parallel for schedule(dynamic)
	#endif
	for (int i=0; i<m_dbParts; i++)
	{
		size_t numBuckets = m_partPointer[i+1]-m_partPointer[i];
		m_partSize[i] = (numBuckets + 1) * sizeof(uint32_t);
		hipHostMalloc(&h_bucketPointers[i], m_partSize[i]);
		CUERR
		
		h_bucketPointers[i][0] = 0;
		
		// partial_sum
		size_t firstIndex = m_partPointer[i];
		size_t lastIndex = m_partPointer[i+1];
		uint32_t* result = h_bucketPointers[i]+1;

		if (firstIndex < lastIndex)
		{
		  uint32_t val = 0;
		  if(choice[firstIndex] == 2)
		  {
			val = bucketSizes[firstIndex];
		  }
		  *result = val;
		  while (++firstIndex < lastIndex)
		  {
			if(choice[firstIndex] == 2)
			{
				val = val + bucketSizes[firstIndex];
			}
			if (val < *result)
			{
				std::cerr << "Bucket pointer overflow. Abort.\n";
			}
			*++result = val;
		  }
		  ++result;
		}
		
		m_partSizeKeys[i]   = h_bucketPointers[i][numBuckets] * sizeof(HKMERr);
		m_partSizeLabels[i] = h_bucketPointers[i][numBuckets] * sizeof(ILBL);

		//~ std::cerr << i+1 << "/" << (int)m_dbParts << " database: " << h_bucketPointers[i][numBuckets] << " elements, "
				  //~ << (m_partSize[i]+m_partSizeKeys[i]+m_partSizeLabels[i])/1000/1000.0 << " MB\n";
		
		//~ std::cerr << "Pointers: " << m_partSize[i]/1000/1000.0
				  //~ << " MB\t Keys: " << m_partSizeKeys[i]/1000/1000.0
				  //~ << " MB\t Labels: " << m_partSizeLabels[i]/1000/1000.0
				  //~ << " MB\n";
	}
	
	// point to db parts in keys and labels
	m_partPointerKeys.resize(m_dbParts+1);
	m_partPointerKeys[0] = 0;
	
	std::vector<size_t>	max_partSize(m_numDevices*m_dbPartsPerDevice,0),
						max_partSizeKeys(m_numDevices*m_dbPartsPerDevice,0),
						max_partSizeLabels(m_numDevices*m_dbPartsPerDevice,0);	
	
	for (int i=0; i<m_dbParts; i++)
	{
		size_t numBuckets = m_partPointer[i+1]-m_partPointer[i];
		m_partPointerKeys[i+1] = m_partPointerKeys[i] + h_bucketPointers[i][numBuckets];
		
		int device = (i/m_dbPartsPerDevice) % m_numDevices;
		//~ std::cerr << "max size - Device " << device << " Index " << i << std::endl;
		if (max_partSize[device] 		< m_partSize[i]) 		max_partSize[device] 		= m_partSize[i];
		if (max_partSizeKeys[device]  	< m_partSizeKeys[i]) 	max_partSizeKeys[device] 	= m_partSizeKeys[i];
		if (max_partSizeLabels[device]  < m_partSizeLabels[i]) 	max_partSizeLabels[device] 	= m_partSizeLabels[i];
	}
	//~ for (int i=0; i<m_dbParts; i++)
	//~ {
		//~ std::cerr << max_partSize[i] << " " << max_partSizeKeys[i] << " " << max_partSizeLabels[i] << "\n";
	//~ }
	
	#ifdef _OPENMP
	#pragma omp parallel
	#endif
	{	
		/// read bucket contents
		#ifdef _OPENMP
		#pragma omp single nowait
		#endif
		{
			for (int i=0; i<m_dbParts; i++)
			{				
				hipHostAlloc(&h_keys[i], m_partSizeKeys[i], 0);
				//~ hipHostAlloc(&h_labels[i], m_partSizeLabels[i], 0);
				CUERR
				
				//~ std::cerr << "Reading...\n";
				if (_modCollision <= 1)
				{	// read everything
					ifs_key.read((char*) &h_keys[i][0], m_partSizeKeys[i]);
					//~ ifs_lbl.read((char*) &h_keys[i][0], m_partSizeLabels[i]);
				}
				else
				{	// read if choice == 2
					uint8_t bucketSize;
					size_t ignoreSize = 0;
					//~ char keybuffer[10241];
					//~ ifs_key.rdbuf()->pubsetbuf(keybuffer,10241);
					uint32_t storeIndexKeys = 0;
					// ~ uint32_t storeIndexLabels = 0;
					for (uint32_t j = m_partPointer[i]; j < m_partPointer[i+1]; j++)
					{
						bucketSize = bucketSizes[j];
						switch(choice[j])
						{
							case 1:
								ignoreSize += bucketSize;
								// skip
								//~ ifs_key.ignore(bucketSize*sizeof(HKMERr));
								//~ ifs_lbl.ignore(bucketSize*sizeof(ILBL));
								break;
							case 2:
								//skip
								ifs_key.ignore(ignoreSize*sizeof(HKMERr));
								//~ ifs_lbl.ignore(ignoreSize*sizeof(ILBL));
								ignoreSize = 0;
								// read
								ifs_key.read((char*) &h_keys[i][storeIndexKeys], bucketSize*sizeof(HKMERr));
								storeIndexKeys += bucketSize;
								// ~ ifs_lbl.read((char*) &h_labels[storeIndexLabels], bucketSize*sizeof(ILBL));
								// ~ storeIndexLabels += bucketSize;
								break;
						}
					}
				}
			}
		}
		#ifdef _OPENMP
		#pragma omp single
		#endif
		{
			for (int i=0; i<m_dbParts; i++)
			{
				// ~ hipHostAlloc(&h_keys[i], m_partSizeKeys[i], 0);
				hipHostAlloc(&h_labels[i], m_partSizeLabels[i], 0);
				CUERR
			
				//~ std::cerr << "Reading...\n";
				if (_modCollision <= 1)
				{	// read everything
					//~ ifs_key.read((char*) &h_keys[i][0], m_partSizeKeys[i]);
					ifs_lbl.read((char*) &h_labels[i][0], m_partSizeLabels[i]);
				}
				else
				{	// read if choice == 2
					uint8_t bucketSize;
					size_t ignoreSize = 0;				
					//~ char lblbuffer[10241];	
					//~ ifs_lbl.rdbuf()->pubsetbuf(lblbuffer,10241);
					//~ uint32_t storeIndexKeys = 0;
					uint32_t storeIndexLabels = 0;
					for (uint32_t j = m_partPointer[i]; j < m_partPointer[i+1]; j++)
					{
						bucketSize = bucketSizes[j];
						switch(choice[j])
						{
							case 1:
								ignoreSize += bucketSize;
								// skip
								//~ ifs_key.ignore(bucketSize*sizeof(HKMERr));
								//~ ifs_lbl.ignore(bucketSize*sizeof(ILBL));
								break;
							case 2:
								// skip
								//~ ifs_key.ignore(ignoreSize*sizeof(HKMERr));
								ifs_lbl.ignore(ignoreSize*sizeof(ILBL));
								ignoreSize = 0;
								// read
								//~ ifs_key.read((char*) &h_keys[i][storeIndexKeys], bucketSize*sizeof(HKMERr));
								//~ storeIndexKeys += bucketSize;
								ifs_lbl.read((char*) &h_labels[i][storeIndexLabels], bucketSize*sizeof(ILBL));
								storeIndexLabels += bucketSize;
								break;
						}
					}
				}
			}
		}

		#ifdef _OPENMP
		#pragma omp single
		#endif	
		for (int i=0; i<m_numDevices; i++)
		{
			hipSetDevice(i);
			
			for(int j=0; j<m_dbPartsPerDevice; ++j)
			{
				int index = m_dbPartsPerDevice*i+j;
				//~ std::cerr << "alloc - Device " << i << " Index " << index << std::endl;

				// allocate device memory
				hipMalloc(&d_bucketPointers[index], max_partSize[i]);
				CUERR
				hipMalloc(&d_keys[index], max_partSizeKeys[i]);
				CUERR
				hipMalloc(&d_labels[index], max_partSizeLabels[i]);
				CUERR
			}
		}
 	}
 	std::cerr << "DB loaded in RAM.\n";
     
	return true;
}

/**
 *  Swap to next database parts on all GPUs
 */
template <typename HKMERr>
bool CuClarkDB<HKMERr>::swapDbParts ()
{
	if (m_cyclesToDo == 0)
	{
		// reset for possible next file
		m_cyclesToDo = m_cyclesPerDevice;
		return false;
	}
	
	//~ if (m_cyclesToDo < m_cyclesPerDevice)
		//~ std::cerr << "Swapping DB part. ";
	
	int offset = m_numDevices*m_dbPartsPerDevice*(m_cyclesPerDevice - m_cyclesToDo);
	//~ std::cerr << "Offset: " << offset << "\n";
	
	//~ #ifdef _OPENMP
	//~ #pragma omp parallel for
	//~ #endif
	for (int i=0; i<m_numDevices; i++)
	{
		hipSetDevice(i);
		//~ hipDeviceSynchronize();
		//~ CUERR
		for(int j=0; j<m_dbPartsPerDevice; ++j)
		{
			int index = m_dbPartsPerDevice*i+j;
			//~ std::cerr << "Swap - Device " << i << " Index " << index << " Offset " << offset << std::endl;
			
			// copy database to part device
			hipMemcpyAsync(d_bucketPointers[index], &h_bucketPointers[index+offset][0], m_partSize[index+offset], hipMemcpyHostToDevice, 0);
			//~ CUERR
			hipMemcpyAsync(d_keys[index],   &h_keys  [index+offset][0], m_partSizeKeys[index+offset],   hipMemcpyHostToDevice, 0);
			//~ CUERR
			hipMemcpyAsync(d_labels[index], &h_labels[index+offset][0], m_partSizeLabels[index+offset], hipMemcpyHostToDevice, 0);
			//~ CUERR
		}
		
		//~ hipDeviceSynchronize();
		//~ CUERR
	}
	//~ if (m_cyclesToDo < m_cyclesPerDevice)
		//~ std::cerr << "DB parts to do: " << m_cyclesToDo << std::endl;
	
	m_cyclesToDo--;
	return true;
}

/**
 *  Set all parameters for a batch
 */
template <typename HKMERr>
bool CuClarkDB<HKMERr>::readyBatch (const size_t _batchId, const size_t _numReads, const size_t _containerCount)
{
	m_numReads[_batchId] = _numReads;
	m_sizeReadsPointer[_batchId] = (_numReads+1)*sizeof(uint32_t);
	m_sizeReadsInContainers[_batchId] = _containerCount*sizeof(CONTAINER);

	return true;
}

/**
 * 	Schedule a prepared batch for database query on GPUs,
 *  Schedule results merging if needed,
 * 	Schedule calculation of final results at the end.
 */
template <typename HKMERr>
bool CuClarkDB<HKMERr>::queryBatch (const size_t _batchId, const bool _isExtended, const bool _isFollowup)
{
	size_t d_pitch = m_sizeResultRow;
	
	hipStream_t stream = 0;
    //~ hipStreamCreate(&stream);
    
	for (int i=0; i<m_numDevices; i++)
	{
		hipSetDevice(i);
		hipMemcpyAsync(d_readsPointer[i], h_readsPointer[_batchId], m_sizeReadsPointer[_batchId], hipMemcpyHostToDevice, stream);
		//~ CUERR
		hipMemcpyAsync(d_readsInContainers[i], h_readsInContainers[_batchId], m_sizeReadsInContainers[_batchId], hipMemcpyHostToDevice, stream);
		//~ CUERR
	}
	
	// process one read per block
	size_t numBlocks = m_numReads[_batchId];
	
	int dbPartOffset = m_numDevices*m_dbPartsPerDevice*(m_cyclesPerDevice - m_cyclesToDo -1);
	//~ std::cerr << "DB Offset: " << dbPartOffset << "\n";
	
	for (int i=0; i<m_numDevices; i++)
	{
		hipSetDevice(i);
		
		for(int j=0; j<m_dbPartsPerDevice; ++j)
		{
			int index = m_dbPartsPerDevice*i+j;
			//~ if(_batchId==0) std::cerr << "query - Device " << i << " Index " << index << " Offset " << dbPartOffset << std::endl;
			queryKernel<<<numBlocks, m_threadsPerBlock_queryKernel, m_sharedSize_queryKernel, stream>>>
							(m_k,
							d_readsPointer[i], d_readsInContainers[i],
							d_bucketPointers[index], d_keys[index], d_labels[index],
							m_partPointer[index+dbPartOffset], m_partPointer[index+dbPartOffset+1],
							d_results[i][j], d_pitch, m_numTargets);
			//~ hipStreamSynchronize(stream);
			//~ CUERR
		}
	}
		
	size_t 	threadsPerBlock = 1024;		
	numBlocks = (m_numReads[_batchId]-1) / threadsPerBlock +1;
	
	// merge results from different db parts on same device
	for (int i=0; i<m_numDevices; i++)
	{
		hipSetDevice(i);
		
		for(int j=1; j<m_dbPartsPerDevice; j<<=1)
		{
			for (int k=0; j+k<m_dbPartsPerDevice; k+=2*j)
			{
				//~ if(_batchId==0) std::cerr << "merge - Device " << i << std::endl;

				mergeKernel<<<numBlocks,threadsPerBlock,0,stream>>>(d_results[i][k], d_results[i][k+j], d_pitch, m_numReads[_batchId], d_results[i][m_dbPartsPerDevice]);
				//~ hipStreamSynchronize(stream);
				//~ CUERR
				
				// swap pointers so that merged result is first
				RESULTS* dummy = d_results[i][k];
				d_results[i][k] = d_results[i][m_dbPartsPerDevice];
				d_results[i][m_dbPartsPerDevice] = dummy;
			}
		}
	}	
	
	// merge results from different devices
	for (int i=1; i<m_numDevices; i<<=1)
	{
		for (int j=0; j+i<m_numDevices; j+=2*i)
		{
			hipSetDevice(j);
			// sync devices, async to host
			hipMemcpyPeer(d_results[j][1], j, d_results[j+i][0], j+i, m_sizeResultRow*m_numReads[_batchId]);	
			// async to other streams
			//~ hipMemcpyPeerAsync(d_results[2], 0, d_results[1], 1, m_sizeResultRow*m_numReads[_batchId], stream);	
			
			mergeKernel<<<numBlocks,threadsPerBlock,0,stream>>>(d_results[j][0], d_results[j][1], d_pitch, m_numReads[_batchId], d_results[j][2]);
			//~ hipStreamSynchronize(stream);
			//~ CUERR
			
			// swap pointers so that merged result is first
			RESULTS* dummy = d_results[j][0];
			d_results[j][0] = d_results[j][2];
			d_results[j][2] = dummy;			
		}
	}
	
	hipSetDevice(0);
	// merge with previous results
	if(_isFollowup)
	{
		//~ std::cerr << "Batch " << _batchId << " retrieve from host" << std::endl;
		
		hipMemcpyAsync(d_results[0][1], h_results[_batchId], m_sizeResultRow*m_numReads[_batchId], hipMemcpyHostToDevice, stream);
				
		mergeKernel<<<numBlocks,threadsPerBlock,0,stream>>>(d_results[0][0], d_results[0][1], d_pitch, m_numReads[_batchId], d_results[0][2]);
		//~ hipStreamSynchronize(stream);
		//~ CUERR
		
		// swap pointers so that merged result is first
		RESULTS* dummy = d_results[0][0];
		d_results[0][0] = d_results[0][2];
		d_results[0][2] = dummy;		
	}
	
	// copy result to host
	if (m_cyclesToDo > 0 || _isExtended)
	{	// copy full results to host
		//~ std::cerr << "Batch " << _batchId << " copy to host" << std::endl;
				
		hipMemcpyAsync(h_results[_batchId], d_results[0][0], m_sizeResultRow*m_numReads[_batchId], hipMemcpyDeviceToHost, stream);
		//~ hipStreamSynchronize(stream);
		//~ CUERR

		//~ std::cerr << "Result size: " << m_sizeResultRow*_numReads / 1000000.0 << " MB" << std::endl;
	}
	
	if (m_cyclesToDo == 0)
	{	// calculate final results and copy to host
		//~ std::cerr << "Batch " << _batchId << " calculate results" << std::endl;
		
		resultKernel<<<numBlocks,threadsPerBlock,0,stream>>>(d_results[0][0], d_pitch, m_numReads[_batchId], d_resultsFinal[0], m_sizeResultFinalRow);
		//~ hipStreamSynchronize(stream);
		//~ CUERR
		
		hipMemcpyAsync(h_resultsFinal[_batchId], d_resultsFinal[0], m_sizeResultFinalRow*m_numReads[_batchId], hipMemcpyDeviceToHost, stream);
		//~ hipStreamSynchronize(stream);
		//~ CUERR
		
		//~ std::cerr << "Result size: " << m_sizeResultFinalRow*m_numReads[_batchId] / 1000000.0 << " MB" << std::endl;

		hipEventRecord(m_batchFinishedEvents[_batchId], stream);
		return true;
	}
	
	//~ hipStreamSynchronize(stream);
	//~ CUERR	
	//~ hipDeviceSynchronize();
	//~ CUERR
	return false;
}

/**
 * Get the two targets with highest scores and sum of all targets.
 */
template <typename HKMERr>
bool CuClarkDB<HKMERr>::getFinalResult (const size_t _batchId, RESULTS* _resutlsFinal)
{
	size_t d_pitch = m_sizeResultRow;
	
	int device = _batchId % m_numDevices;
	hipSetDevice(device);
	
	hipMemcpyAsync(d_results[device][0], h_results[_batchId], m_sizeResultRow*m_numReads[_batchId], hipMemcpyHostToDevice, 0);
	
	size_t threadsPerBlock = 1024;
	size_t numBlocks = (m_numReads[_batchId]-1) / threadsPerBlock +1;
	//~ std::cerr << "# reads: " << m_numReads[_batchId] << " # blocks: " << numBlocks << "\n";
	resultKernel<<<numBlocks,threadsPerBlock,0,0>>>(d_results[device][0], d_pitch, m_numReads[_batchId], d_resultsFinal[device], m_sizeResultFinalRow);
	
	hipMemcpyAsync(_resutlsFinal, d_resultsFinal[device], m_sizeResultFinalRow*m_numReads[_batchId], hipMemcpyDeviceToHost, 0);
	
	hipEventRecord(m_batchFinishedEvents[_batchId], 0);
	
	return true;
}


/**
 * Queries a batch against a database part.
 * 
 * Processes one read per block:
 * Loads read data into shared memory,
 * contructs one kmer per thread and queries it,
 * scores in shared memory,
 * continues until the read is completed.
 * Stores non zero scores in global memory.
 */
template <typename HKMERr>
__global__ void queryKernel (uint8_t k,
			uint32_t* readsPointer, CONTAINER* readsInContainers,
			uint32_t* bucketPointers, HKMERr* keys, ILBL* labels,
			uint32_t dbPartStart, uint32_t dbPartEnd,
			RESULTS* results, size_t pitch, size_t numTargets)
{
	int tid = threadIdx.x;
	int bid = blockIdx.x;
	int wid = threadIdx.x/warpSize;
	int wlane = threadIdx.x % warpSize;
	int numWarps = blockDim.x/warpSize;
	size_t nucsPerCon = sizeof(CONTAINER)*4;
	
	// get row (=bid) to store results
	RESULTS* resultRow = (RESULTS*) ((char*)results + bid*pitch);
	//~ uint32_t* resultRow = &results[bid*numTargets];
	
	// shared memory for scoring
	extern __shared__ uint32_t targetHits[];
	uint16_t* targetHits16 = (uint16_t*) &targetHits[0];
	RESULTS* sharedResultRow = (RESULTS*) &targetHits16[((numTargets-1)/2+1)*2];
	CONTAINER* sharedContainers = (CONTAINER*) &sharedResultRow[0];
	
	// set all target counters to zero
	for (int i=tid; i<numTargets; i += blockDim.x)
	{
		targetHits16[i] = 0;
	}
	__syncthreads();	// for blockDim > warp size
	
	uint64_t kmer;
	ILBL target;
	uint64_t cutoff = (uint64_t)-1 >> (64 - 2*k);
	
	CONTAINER partLength, offset, tmp;	
	uint32_t partPointer = readsPointer[bid];
	uint32_t readEnd = readsPointer[bid+1];
	//~ uint32_t partBegin;
	uint32_t partIterator;
	short numKmer;
	
	uint32_t firstContainer;
	uint32_t containerPerWarp = (k+warpSize-2)/nucsPerCon+1;
	//~ if(tid==0) printf("Block %d: MaxCon %u\n",bid,maxContainers);
	
	while(partPointer < readEnd)
	{
		partLength = readsInContainers[partPointer];
		//~ partBegin = ++partPointer + tid / nucsPerCon;
		firstContainer = ++partPointer;
		partPointer += (partLength-1) / nucsPerCon +1;
		// number of kmer for wlane == 0 to check if warp has work to do
		numKmer = partLength - k + 1 - wid*warpSize;
		while (numKmer > 0)
		{
			// load containers for own warp
			if(wlane < containerPerWarp)
			{
				int readIndex = firstContainer+wid*warpSize/nucsPerCon+wlane;
				if (readIndex < partPointer)
					sharedContainers[wid*containerPerWarp + wlane] 
						= readsInContainers[readIndex];
			}
			// check if thread has work to do
			if(numKmer-wlane <= 0) break;
			partIterator = wid*containerPerWarp + wlane / nucsPerCon;	// shared version
			//~ partIterator = firstContainer + tid / nucsPerCon;	// global version
			
			kmer = 0;
			// read full containers
			for (int i=0; i<(k+tid%nucsPerCon)/ nucsPerCon; ++i)
			{
				kmer <<= 2*nucsPerCon;
				kmer |= sharedContainers[partIterator++];	// shared version
				//~ kmer |= readsInContainers[partIterator++];	// global version
			}
			// read 'offset' additional nucs
			offset = (k+tid) % nucsPerCon;
			if (offset != 0)
			{				
				kmer <<= 2*offset;
				tmp = sharedContainers[partIterator];	// shared version
				//~ tmp = readsInContainers[partIterator];	// global version
				tmp >>= 2*(nucsPerCon-offset);
				kmer |= tmp;
			//~ if(tid==warpSize-1) printf("Block %d: Iterator %u\n",bid,partIterator);
			}
			// cut off overhang
			kmer &= cutoff;
			
			// print for debugging
			/*
			printf("Block %d, Thread %2d: %" PRIu64 "\n",bid,tid,kmer);
			uint64_t x = 3;
			char kmer_string[28];
			kmer_string[27] = '\0';
			for(int j=0; j<k; ++j)
			{
				switch ( (kmer & x) >> (2*j) )
				{
					case(0): kmer_string[k-j-1] = 'T'; break;
					case(1): kmer_string[k-j-1] = 'G'; break;
					case(2): kmer_string[k-j-1] = 'C'; break;
					case(3): kmer_string[k-j-1] = 'A'; break;
				}
				x <<= 2;
			}
			printf("Block %d, Thread %2d: %s\n",bid,tid,kmer_string);
			*/
			
			if (queryElement(k, kmer, bucketPointers, keys, labels, dbPartStart, dbPartEnd, target))
			{
				// 32bit targetHits version
				//~ atomicAdd(&targetHits[target], 1);
				//~ printf("Target: %d\n",target);
				
				// 16bit targetHits version
				ILBL target32 = target / 2;
				//~ uint32_t value = target % 2 ? 1<<16 : 1;
				uint32_t value = 1 <<((target % 2)*16);
				atomicAdd(&targetHits[target32],value);
				//~ printf("Block: %d, Target %d, counter %d\n",bid, target,targetHits16[target]);
				//~ printf("Target: %d, target32: %d, value %d, counter: %d\n",target,target32,value,targetHits16[target]);
			}
			// advance to next section
			numKmer -= blockDim.x;
			firstContainer += blockDim.x/nucsPerCon;
		}				
	}
	__syncthreads();	// for blockDim > warp size
	
	// store nonzeros with index in global
	// cf. http://www.davidespataro.it/cuda-stream-compaction-efficient-implementation/
	//~ if (wid == 0)
	{
		int pred, t_m, b, t_u, total=0;
		int j;
		size_t numTargetsPerWarp = (numTargets-1) / numWarps +1;
		numTargetsPerWarp = ((numTargetsPerWarp-1) / warpSize +1) * warpSize;
		//~ if(wlane==0) printf("numTargetsPerWarp: %d\n",numTargetsPerWarp);
		//~ for (int i=tid; i<numTargets; i += warpSize)
		for (int i=wlane+numTargetsPerWarp*wid; (i<numTargetsPerWarp*(wid+1)) && (i <numTargets) ; i += warpSize)
		{
			pred = targetHits16[i] > 0 ? 1 : 0;
			t_m = INT_MAX >> warpSize-wlane-1;	// set bits < tid
			b = __ballot(pred) & t_m;			// get pred bits < tid
			t_u = __popc(b);					// get sum of bits = # pred < tid
			j = 2*(total+t_u);

			if (pred)
			{
				if (j+2 < pitch)
				{
					sharedResultRow[j+1+wid*(pitch/sizeof(RESULTS))] = i;
					sharedResultRow[j+2+wid*(pitch/sizeof(RESULTS))] = targetHits16[i];
					//~ printf("Block: %i, Target: %i, Hits: %i\n",bid,i,targetHits[i]);
				}
				else
				{
					printf("Too many different tagets hit by a sequence. Results will be corrupted.\n");
				}
			}
			total += t_u+pred;
			if (i == numTargetsPerWarp*(wid+1)-1 || i == numTargets-1)
			{
				//~ printf("Block: %i, Total: %i\n",bid,total);
				sharedResultRow[0+wid*(pitch/sizeof(RESULTS))] = total;
			}
			total = __shfl(total, warpSize-1);	// get total from last lane
		}
		

	}
	__syncthreads();	// for blockDim > warp size	
	
	int subtotal = 0;
	int warpTotal = sharedResultRow[wid*(pitch/sizeof(RESULTS))];
	for(int j=0; j<wid; j++)
	{
		subtotal += sharedResultRow[j*(pitch/sizeof(RESULTS))];	
	}
	for (int i=wlane; i<warpTotal*2; i+= warpSize)
			resultRow[i+subtotal*2+1] = sharedResultRow[i+wid*(pitch/sizeof(RESULTS))+1];
	if(wid==numWarps-1 && wlane == 0)
	{
		resultRow[0] = subtotal+warpTotal;
		//~ printf("Block: %d, Targets hit: %d\n",bid,subtotal+warpTotal);
	}
}

/**
 *  Query k-mer against a database part.
 *  Analog to hashTable_hh find, for canonical kmer
 */
template <typename HKMERr>
__device__ bool queryElement (const uint8_t& k, const uint64_t& _ikmer,
		uint32_t* d_bucketPointers, HKMERr* d_keys, ILBL* d_labels,
		uint32_t dbPartStart, uint32_t dbPartEnd,
		ILBL& _returnLabel)
{
	// getting reverse kmer
	size_t _ikmerR = _ikmer;
	// The following 6 lines come from Jellyfish source code
	_ikmerR = ((_ikmerR >> 2)  & 0x3333333333333333UL) | ((_ikmerR & 0x3333333333333333UL) << 2);
	_ikmerR = ((_ikmerR >> 4)  & 0x0F0F0F0F0F0F0F0FUL) | ((_ikmerR & 0x0F0F0F0F0F0F0F0FUL) << 4);
	_ikmerR = ((_ikmerR >> 8)  & 0x00FF00FF00FF00FFUL) | ((_ikmerR & 0x00FF00FF00FF00FFUL) << 8);
	_ikmerR = ((_ikmerR >> 16) & 0x0000FFFF0000FFFFUL) | ((_ikmerR & 0x0000FFFF0000FFFFUL) << 16);
	_ikmerR = ( _ikmerR >> 32                        ) | (_ikmerR                        << 32);
	_ikmerR = (((uint64_t)-1) - _ikmerR) >> (64 - (k << 1));
	
	// getting canonical kmer
	size_t _ikmerC = _ikmer < _ikmerR ? _ikmer : _ikmerR;
	
	size_t quotient = _ikmerC / HTSIZE;
	size_t remainder = _ikmerC - quotient * HTSIZE;
	
	// check for correct dbPart
	if (remainder < dbPartStart || remainder >= dbPartEnd)
		return false;
	remainder -= dbPartStart;
	
	//~ if(dbPart==1)
		//~ printf("Part: %d, Kmer: %" PRIu64 ", Remainder: %" PRIu64 "\n", dbPart, _ikmerC, remainder);

	size_t bucketBegin = d_bucketPointers[remainder];
	size_t bucketEnd = d_bucketPointers[remainder+1];
	//~ if(dbPart==1)
		//~ printf("Bucket size: %2d, Remainder: %" PRIu64 "\n", bucketEnd-bucketBegin, remainder-HTSIZE/2);

	if(	bucketEnd-bucketBegin > 0)
	{	// bucket not empty	
		size_t i = bucketBegin;
		HKMERr key = d_keys[i];
		
		if( key > quotient || d_keys[bucketEnd-1] < quotient)
		{	// quotient not in range
			return false;
		}
		
		while(key <= quotient)
		{
			if(key == quotient)
			{	// key found
				_returnLabel = d_labels[i];
				//printf("Part: %d, Label: %4d, Remainder: %8d\n", dbPart, _returnLabel, remainder);
				return true;
			}
			key = d_keys[++i];
		}
		// key not in list
		return false;
	} 

	// bucket empty
	return false;
}

/**
 * Merge two results into one.
 * 1 thread handles results of 1 read.
 * cf. https://nvlabs.github.io/moderngpu/merge.html
 */
__global__ void mergeKernel (RESULTS* resultA, RESULTS* resultB, size_t pitch, size_t numReads, RESULTS* results)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	
	if (tid < numReads)
	{
		RESULTS* resultRowA = (RESULTS*) ((char*)resultA + tid*pitch);
		RESULTS* resultRowB = (RESULTS*) ((char*)resultB + tid*pitch);
		RESULTS* resultsRow = (RESULTS*) ((char*)results + tid*pitch);
		
		// get number of results
		RESULTS countA = resultRowA[0];
		RESULTS countB = resultRowB[0];
		int count = countA + countB;
		//~ printf("Thread: %d, CountA: %d, CountB: %d\n",tid,countA,countB);
		if (count > 4*MAXHITS+4)
		{
			printf("Count overflow\n");
			count = 0;
		}
		
		int indexA = 1;
		int indexB = 1;
		
		// get target ids of first results
		RESULTS targetA = resultRowA[indexA];
		RESULTS targetB = resultRowB[indexB];
		
		//~ for(int i=0; i<countA; ++i)
		//~ {
			//~ printf("Thread: %d, targetA: %d\n",tid,resultRowA[2*i+1]);
		//~ }
		//~ for(int i=0; i<countB; ++i)
		//~ {
			//~ printf("Thread: %d, targetB: %d\n",tid,resultRowB[2*i+1]);
		//~ }
		
		// merge results
		for(int i=0; i<count; ++i)
		{
			//~ printf("Thread: %d, targetA: %d, targetB: %d\n",tid,targetA,targetB);
			
			// find next result
			uint8_t choice;
			// choice 0 -> A
			// choice 1 -> B
			// choice 2 -> A=B, merge
			
			if (indexB > countB*2)		// B done?
				choice = 0;	
			else if (indexA > countA*2)	// A done?
				choice = 1;
			else if (targetA < targetB) // A smaller?
				choice = 0;
			else if (targetA > targetB)	// B smaller?
				choice = 1;
			else						// same value
				choice = 2;
			
			// put target id
			resultsRow[2*i+1] = choice ? targetB : targetA;
			// put target counter
			resultsRow[2*i+2] = (choice ? resultRowB[++indexB] : resultRowA[++indexA])
							+ (choice == 2 ? resultRowA[++indexA] : 0);
			
			// get next target ids
			switch(choice)
			{	
				case 0:
					targetA = resultRowA[++indexA];
					break;
				case 1:
					targetB = resultRowB[++indexB];
					break;
				case 2:
					targetA = resultRowA[++indexA];
					targetB = resultRowB[++indexB];
					// one less output because of sum
					count--;
					break;				
			}
		}
		// put number of results
		resultsRow[0] = count;

		//~ for(int i=0; i<count; ++i)
			//~ printf("Thread: %d, target: %d, hits: %d\n",tid,resultsRow[2*i+1],resultsRow[2*i+2]);
	}
}

/**
 * Find best, second best and sum of scores for each read.
 * 1 thread handles results of 1 read.
 */
__global__ void resultKernel (RESULTS* scores, size_t spitch, size_t numReads, RESULTS* results, size_t rpitch)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	
	if (tid < numReads)
	{
		RESULTS* scoresRow = (RESULTS*) ((char*)scores + tid*spitch);
		RESULTS* resultsRow = (RESULTS*) ((char*)results + tid*rpitch);
		
		RESULTS best = 0, s_best = 0;
		RESULTS indexBest = 0, index_sBest = 0;
		RESULTS sumN = 0;
		
		RESULTS count = scoresRow[0];
		//~ printf("Read: %d, count: %d\n", tid, count);
		
		RESULTS targetScore;
		
		for(int i=0; i<count; ++i)
		{
			targetScore = scoresRow[2*i+2];
			
			// new best, update best and second best
			if (targetScore > best)
			{
				s_best = best;
				index_sBest = indexBest;
				best = targetScore;
				indexBest = scoresRow[2*i+1] + 1;					
			}
			// new second best, update
			else if (targetScore > s_best)
			{
				s_best = targetScore;
				index_sBest = scoresRow[2*i+1] + 1;
			}
			sumN += targetScore;
		}
		
		resultsRow[0] = sumN;
		resultsRow[1] = indexBest;
		resultsRow[2] = best;
		resultsRow[3] = index_sBest;
		resultsRow[4] = s_best;
		
		//~ printf("Thread: %d, sum: %d, best: %d, sbest: %d\n",tid,sumN,indexBest,index_sBest);
	}
}

// instantiations
template class CuClarkDB<uint16_t>;
template class CuClarkDB<uint32_t>;
template class CuClarkDB<uint64_t>;
